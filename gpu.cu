#include "hip/hip_runtime.h"
#include "common.h"
#include <cmath>
#include <cstring>
#include <iostream>
#include <hip/hip_runtime.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/scan.h>

#define NUM_THREADS 256

// Put any static global variables here that you will use throughout the simulation.
int blks;
int grid_size, num_bins;
double cell_size;
int *d_bin_count, *d_bin_ids, *d_part_ids;

__device__ void apply_force_gpu(particle_t& particle, particle_t& neighbor) {
    double dx = neighbor.x - particle.x;
    double dy = neighbor.y - particle.y;
    double r2 = dx * dx + dy * dy;
    if (r2 > cutoff * cutoff)
        return;
    // r2 = fmax( r2, min_r*min_r );
    r2 = (r2 > min_r * min_r) ? r2 : min_r * min_r;
    double r = sqrt(r2);

    //
    //  very simple short-range repulsive force
    //
    double coef = (1 - cutoff / r) / r2 / mass;
    particle.ax += coef * dx;
    particle.ay += coef * dy;
}

__global__ void compute_forces_gpu(particle_t* particles, int num_parts) {
    // Get thread (particle) ID
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid >= num_parts)
        return;

    particles[tid].ax = particles[tid].ay = 0;
    for (int j = 0; j < num_parts; j++)
        apply_force_gpu(particles[tid], particles[j]);
}

__global__ void move_gpu(particle_t* particles, int num_parts, double size) {

    // Get thread (particle) ID
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid >= num_parts)
        return;

    particle_t* p = &particles[tid];
    //
    //  slightly simplified Velocity Verlet integration
    //  conserves energy better than explicit Euler method
    //
    p->vx += p->ax * dt;
    p->vy += p->ay * dt;
    p->x += p->vx * dt;
    p->y += p->vy * dt;

    //
    //  bounce from walls
    //
    while (p->x < 0 || p->x > size) {
        p->x = p->x < 0 ? -(p->x) : 2 * size - p->x;
        p->vx = -(p->vx);
    }
    while (p->y < 0 || p->y > size) {
        p->y = p->y < 0 ? -(p->y) : 2 * size - p->y;
        p->vy = -(p->vy);
    }
}

void init_simulation(particle_t* parts, int num_parts, double size) {
    // You can use this space to initialize data objects that you may need
    // This function will be called once before the algorithm begins
    // parts live in GPU memory
    // Do not do any particle simulation here

    blks = (num_parts + NUM_THREADS - 1) / NUM_THREADS;

    grid_size = static_cast<int>(std::floor(size / cutoff)); // number of bins on one side
    num_bins = grid_size * grid_size; // number of bins
    cell_size = size / grid_size; // size of a bin

    // allocate memory on GPU
    hipMalloc((void**)&d_bin_count, num_bins * sizeof(int));
    hipMalloc((void**)&d_bin_ids, num_bins * sizeof(int));
    hipMalloc((void**)&d_part_ids, num_parts * sizeof(int));
}

// CUDA, zero accelerations
__global__ void zero_accelerations(particle_t* parts, int num_parts) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < num_parts) {
        parts[idx].ax = 0;
        parts[idx].ay = 0;
    }
}

// CUDA, compute bin counts
__global__ void compute_bin_counts(particle_t* parts, int* bin_count, int num_parts, float cell_size, int grid_size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < num_parts) {
        int cell_col = static_cast<int>(parts[idx].x / cell_size);
        int cell_row = static_cast<int>(parts[idx].y / cell_size);
        atomicAdd(&bin_count[cell_row * grid_size + cell_col], 1);  // Atomic increment
    }
}

// CUDA, bin particles
__global__ void bin_particles(particle_t* parts, int* bin_count, int* bin_ids, int* part_ids, int num_parts, float cell_size, int grid_size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < num_parts) {
        int cell_col = static_cast<int>(parts[idx].x / cell_size);
        int cell_row = static_cast<int>(parts[idx].y / cell_size);
        int bin_idx = cell_row * grid_size + cell_col;

        int pos = atomicAdd(&bin_count[bin_idx], 1);  // Get position in bin and increment
        part_ids[bin_ids[bin_idx] + pos] = idx;  // Store particle ID at correct position
    }
}

void simulate_one_step(particle_t* parts, int num_parts, double size) {
    
    // Reset bin counts
    hipMemset(d_bin_count, 0, num_bins * sizeof(int));
    hipMemset(d_bin_ids, -1, num_bins * sizeof(int));
    hipMemset(d_part_ids, 0, num_parts * sizeof(int));

    zero_accelerations<<<blks, NUM_THREADS>>>(parts, num_parts);
    
    // bin particles
    // acquire bin counts
    compute_bin_counts<<<blks, NUM_THREADS>>>(parts, d_bin_count, num_parts, cell_size, grid_size);

    // compute prefix sums
    thrust::device_ptr<int> dev_bin_count(d_bin_count);
    thrust::device_ptr<int> dev_bin_ids(d_bin_ids);
    thrust::exclusive_scan(dev_bin_count, dev_bin_ids + num_bins, dev_bin_ids);  // Exclusive scan

    // write over all elements in bin_count to 0 
    // hipMemset(d_bin_count, 0, num_bins * sizeof(int));
    
    // binning
    // bin_particles<<<blks, NUM_THREADS>>>(parts, d_bin_count, d_bin_ids, d_part_ids, num_parts, cell_size, grid_size);

    // copy back to CPU for debug
    int *bin_count = new int[num_bins];
    int *bin_ids = new int[num_bins];
    int *part_ids = new int[num_parts];
    hipMemcpy(bin_count, d_bin_count, num_bins * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(bin_ids, d_bin_count, num_bins * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(part_ids, d_bin_count, num_parts * sizeof(int), hipMemcpyDeviceToHost);

    std::cout << "Bin Counts: " << "\n";
    for (int bin_row = grid_size - 1; bin_row >= 0; --bin_row) {
        for (int bin_col = 0; bin_col < grid_size; ++bin_col) {
            std::cout << bin_count[bin_row * grid_size + bin_col] << " ";
        }
        std::cout << "\n";
    }
    std::cout << "Bin IDs: " << "\n";
    for (int bin_row = grid_size - 1; bin_row >= 0; --bin_row) {
        for (int bin_col = 0; bin_col < grid_size; ++bin_col) {
            std::cout << bin_ids[bin_row * grid_size + bin_col] << " ";
        }
        std::cout << "\n";
    }
    std::cout << "Particle IDs: " << "\n";
    for (int part_i = 0; part_i < num_parts; ++part_i) {
        std::cout << part_ids[part_i] << " ";
        std::cout << "\n";
    }

    // COMPUTE FORCES
    
    // interiors

    // left edge 

    // right edge

    // top edge

    // bottom edge

    // left-top corner

    // right-top corner

    // left-bottom corner

    // right-bottom corner 

    // MOVE PARTICLES
    
    // Compute forces
    //compute_forces_gpu<<<blks, NUM_THREADS>>>(parts, num_parts);

    // Move particles
    //move_gpu<<<blks, NUM_THREADS>>>(parts, num_parts, size);
}
